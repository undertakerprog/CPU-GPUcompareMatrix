#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <iomanip>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

using namespace std;

inline size_t idx(size_t r, size_t c, size_t cols) { return r * cols + c; }

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            cerr << "CUDA Error: " << hipGetErrorString(err) \
                 << " at " << __FILE__ << ":" << __LINE__ << endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

#define CURAND_CHECK(call) \
    do { \
        hiprandStatus_t status = call; \
        if (status != HIPRAND_STATUS_SUCCESS) { \
            cerr << "CURAND Error: " << status \
                 << " at " << __FILE__ << ":" << __LINE__ << endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// CUDA kernel: Transpose matrix
// Input: rows x cols -> Output: cols x rows
__global__ void transpose_kernel(const int* src, int* dst, size_t rows, size_t cols) {
    size_t i = blockIdx.y * blockDim.y + threadIdx.y;
    size_t j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < rows && j < cols) {
        // dst[j][i] = src[i][j]
        // src: rows x cols, dst: cols x rows
        size_t src_idx = i * cols + j;
        size_t dst_idx = j * rows + i;
        dst[dst_idx] = src[src_idx];
    }
}

__global__ void float_to_int_kernel(const float* src, int* dst, size_t n, float scale, float shift) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        dst[idx] = (int)(src[idx] * scale + shift);
    }
}

void print_fragment(const int* a, size_t rows, size_t cols, size_t maxr = 10, size_t maxc = 10) {
    size_t rr = min(rows, maxr);
    size_t cc = min(cols, maxc);
    for (size_t i = 0; i < rr; ++i) {
        for (size_t j = 0; j < cc; ++j) {
            cout << setw(6) << a[idx(i,j,cols)];
        }
        cout << "\n";
    }
}

int main(int argc, char** argv) {
    size_t rows = 1024;
    size_t cols = 1024;
    unsigned int seed = (unsigned int) chrono::system_clock::now().time_since_epoch().count();
    bool quiet = false;
    bool save_result = false;
    bool use_curand = true; // use cuRAND

    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "--quiet") == 0) {
            quiet = true;
        } else if (strcmp(argv[i], "--save") == 0) {
            save_result = true;
        } else if (strcmp(argv[i], "--cpu-rand") == 0) {
            use_curand = false;
        } else if (i == 1 && argv[i][0] != '-') {
            rows = stoul(argv[i]);
        } else if (i == 2 && argv[i][0] != '-') {
            cols = stoul(argv[i]);
        } else if (i == 3 && argv[i][0] != '-') {
            seed = (unsigned int) stoi(argv[i]);
        }
    }

    if (!quiet) {
        cout << "GPU Version - Matrix " << rows << " x " << cols << ", seed=" << seed << "\n";
        cout << "Random generator: " << (use_curand ? "cuRAND (GPU)" : "mt19937 (CPU)") << "\n";
    }

    size_t size_input = rows * cols * sizeof(int);
    size_t size_output = cols * rows * sizeof(int);
    
    vector<int> h_A(rows * cols);
    vector<int> h_B(cols * rows);

    // Allocate device memory
    int *d_A, *d_B;
    CUDA_CHECK(hipMalloc(&d_A, size_input));
    CUDA_CHECK(hipMalloc(&d_B, size_output));

    FILE* f_input = fopen("input.bin", "rb");
    bool use_saved_input = (f_input != nullptr) && save_result;
    
    if (use_saved_input) {
        size_t read_count = fread(h_A.data(), sizeof(int), h_A.size(), f_input);
        fclose(f_input);
        
        if (read_count != h_A.size()) {
            cerr << "Error: Failed to read input data from input.bin\n";
            cerr << "Expected " << h_A.size() << " elements, got " << read_count << "\n";
            exit(EXIT_FAILURE);
        }
        
        if (!quiet) {
            cout << "Loaded input data from CPU (input.bin)\n";
        }
        
        CUDA_CHECK(hipMemcpy(d_A, h_A.data(), size_input, hipMemcpyHostToDevice));
        
    } else if (use_curand) {
        float *d_A_float;
        CUDA_CHECK(hipMalloc(&d_A_float, rows * cols * sizeof(float)));

        hiprandGenerator_t gen;
        CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, seed));

        // uniform float [0, 1]
        CURAND_CHECK(hiprandGenerateUniform(gen, d_A_float, rows * cols));

        int threads = 256;
        int blocks = (rows * cols + threads - 1) / threads;
        float_to_int_kernel<<<blocks, threads>>>(d_A_float, d_A, rows * cols, 2000.0f, -1000.0f);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy(h_A.data(), d_A, size_input, hipMemcpyDeviceToHost));

        // cleanup
        CURAND_CHECK(hiprandDestroyGenerator(gen));
        CUDA_CHECK(hipFree(d_A_float));
    } else {
        mt19937 rng(seed);
        uniform_int_distribution<int> dist(-1000, 1000);
        for (size_t i = 0; i < h_A.size(); ++i) h_A[i] = dist(rng);

        CUDA_CHECK(hipMemcpy(d_A, h_A.data(), size_input, hipMemcpyHostToDevice));
    }

    if (!quiet) {
        cout << "\nInput matrix fragment (top-left):\n";
        print_fragment(h_A.data(), rows, cols);
    }

    // Setup grid and block dimensions
    dim3 blockDim(32, 32);  // 1024 threads per block (> 32 как требуется)
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x, 
                 (rows + blockDim.y - 1) / blockDim.y);
    
    if (!quiet) {
        cout << "\nGrid: (" << gridDim.x << ", " << gridDim.y << "), Block: (" 
             << blockDim.x << ", " << blockDim.y << ")\n";
        cout << "Total threads per block: " << (blockDim.x * blockDim.y) << "\n";
        cout << "Total blocks in grid: " << (gridDim.x * gridDim.y) << "\n";
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Warm-up run
    transpose_kernel<<<gridDim, blockDim>>>(d_A, d_B, rows, cols);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Timed run
    CUDA_CHECK(hipEventRecord(start));
    transpose_kernel<<<gridDim, blockDim>>>(d_A, d_B, rows, cols);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float ms_time = 0;
    CUDA_CHECK(hipEventElapsedTime(&ms_time, start, stop));
    float us_time = ms_time * 1000.0f;

    // Copy result back
    CUDA_CHECK(hipMemcpy(h_B.data(), d_B, size_output, hipMemcpyDeviceToHost));

    if (quiet) {
        cout << "GPU: " << (long long)us_time << " us (" << (long long)ms_time << " ms)\n";
    } else {
        cout << "\nTranspose operation:\n";
        cout << "Time: " << (long long)us_time << " us (" << (long long)ms_time << " ms)\n";
        cout << "Output dimensions: " << cols << " x " << rows << "\n";
        cout << "Result fragment (top-left):\n";
        print_fragment(h_B.data(), cols, rows);
        cout << "\nGPU Execution completed successfully!\n";
    }

    if (save_result) {
        FILE* f = fopen("gpu_result.bin", "wb");
        if (f) {
            fwrite(h_B.data(), sizeof(int), h_B.size(), f);
            fclose(f);
        }
    }

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));

    return 0;
}